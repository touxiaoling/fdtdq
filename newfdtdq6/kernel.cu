#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <iostream>
#include <tchar.h>
#include <float.h>
#include <math.h>
#include <fstream>
#include <string>
#include <ctime>
#include "sm_20_atomic_functions.h"
using namespace std;
//������Ƴ���//��Ҫ����ͨ���������룬������ġ�
#define RE 25
#define WE 60
#define JE 120
//Ԫ������
#define TX 16
#define TY 6
#define TZ 4
//block����
#define FREQUENCYRESOLUTION 0.25//Ƶ�ʷֱ���
#define TOTALSIMULATERTIME (1.0/FREQUENCYRESOLUTION)//��ģ�����ʱ��

#define FREQUENCY 50.0
//����Ƶ�ʣ�Hz��

#define NOTHING 1231123
//�絼�ʿ�ѡ���� KNEE AVG NOTHING

//������//��ֱ��ʹ�á�
#define EARTH_RD 6370.0e3
//��m��
#define EARTH_HD 100.0e3
//��m��
#define LIGHT_SPEED 299792458.0
// ����еĹ��٣�m/s��
#define LIGHT_SPEED_SQUARED 89875517873681764.0
// m^2/s^2
#define MU_0 1.2566370614359172953850573533118011536788677597500423283899778369231265625144835994512139301368468271e-6
// ����дŵ��ʣ�H/m��
#define EPSILON_0 8.8541878176203898505365630317107502606083701665994498081024171524053950954599821142852891607182008932e-12
// ����н�糣��(F/m)
#define M_PI 3.141159265358979323846
//����pi��ֵ
//����ʹ�õ�����������DBL_EPSILON��FLT_MAX
#define EPSILON_R 1.0
#define SIGMA 0.0
#define MU_R 1.0
#define SIGMA_M 0.0

int divUp(int a, int b)
{
	return (a + b - 1) / b;
}
float electric(int N, float dt)
{
	float rtau = 0.3 / 20;//Ƶ��
	float tau = rtau / dt;
	return (1000 * exp(-(N - 3 * tau)*(N - 3 * tau) / tau / tau));
}

__global__ void EKernel1(const float*hj, const int rb, const int wb, const int jb, float * sumnum)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	const int z = blockIdx.z*blockDim.z + threadIdx.z;
	const int i = x + y*rb + z*rb*wb;
	const int i0 = x + y*rb + 0 * rb*wb;

	if ((x < rb - 1) && ((y == 0) || (y == wb - 2)) && (z < jb - 1))
	{
		atomicAdd(&(sumnum[x + (y / (wb - 2))*rb]), hj[i]);
	}
}

__global__ void EKernel2(float *er, float*ew, float*ej, const float*hr, const float*hw, const float*hj, const float* ca, const float* cb, const int rb, const int wb, const int jb, const float dr, const float dw, const float dj, const float *sumnum)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	const int z = blockIdx.z*blockDim.z + threadIdx.z;
	const int i = x + y*rb + z*rb*wb;
	const int ix_1 = (x - 1) + y *rb + z*rb*wb;
	const int iy_1 = x + (y - 1)*rb + z*rb*wb;
	const int iz_1 = x + y*rb + (z - 1)*rb*wb;
	const int izje = x + y*rb + (jb - 2)*rb*wb;

	if ((x < rb - 1) && (y == 0) &&  (z < jb))
	{
		er[i] = ca[x] * er[i] + (sin(dw / 2)*dj / (2 * M_PI*(1 - cos(dw / 2))*(EARTH_RD + (x + 0.5)*dr)))*cb[x] * sumnum[x];
	}
	if ((x < rb - 1) && (y == wb - 1) &&  (z < jb))
	{
		er[i] = ca[x] * er[i] - (sin(dw / 2)*dj / (2 * M_PI*(1 - cos(dw / 2))*(EARTH_RD + (x + 0.5)*dr)))*cb[x] * sumnum[x + rb];
	}

	if ((x < rb - 1) && (y > 0) && (y < wb - 1) && (z == 0))
		er[i] = ca[x] * er[i] + cb[x] / (((x + 0.5)*dr + EARTH_RD)*sin(y * dw))*(sin((y + 0.5)*dw)*hj[i] / dw - sin((y - 0.5)*dw)*hj[iy_1] / dw - (hw[i] - hw[izje]) / dj);
	if ((x < rb - 1) && (y > 0) && (y < wb - 1) && (z > 0) && (z < jb - 1))
		er[i] = ca[x] * er[i] + cb[x] / (((x + 0.5)*dr + EARTH_RD)*sin(y * dw))*(sin((y + 0.5)*dw)*hj[i] / dw - sin((y - 0.5)*dw)*hj[iy_1] / dw - (hw[i] - hw[iz_1]) / dj);

	if ((x > 0) && (x < rb - 1) && (y < wb - 1) && (z == 0))
		ew[i] = ca[x + rb] * ew[i] + cb[x + rb] / ((x*dr + EARTH_RD))*((hr[i] - hr[izje]) / (sin((y + 0.5)*dw)*dj) - (x + 0.5 + EARTH_RD / dr)*hj[i] + (x - 0.5 + EARTH_RD / dr)*hj[ix_1]);
	if ((x > 0) && (x < rb - 1) && (y < wb - 1) && (z > 0) && (z < jb - 1))
		ew[i] = ca[x + rb] * ew[i] + cb[x + rb] / ((x*dr + EARTH_RD))*((hr[i] - hr[iz_1]) / (sin((y + 0.5)*dw)*dj) - (x + 0.5 + EARTH_RD / dr)*hj[i] + (x - 0.5 + EARTH_RD / dr)*hj[ix_1]);

	if ((x > 0) && (x < rb - 1) && (y > 0) && (y < wb - 1) && (z < jb - 1))
		ej[i] = ca[x + 2 * rb] * ej[i] + cb[x + 2 * rb] / (((x)*dr + EARTH_RD))*(-((hr[i] - hr[iy_1]) / (dw)-(x + 0.5 + EARTH_RD / dr)*hw[i] + (x - 0.5 + EARTH_RD / dr)*hw[ix_1]));
	//__syncthreads();




}


__global__ void EKernel3(float *er, float*ew, float*ej, const int sr, const int sw, const int sj, const float S, float *sumnum, const int rb, const int wb, const int jb)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	const int z = blockIdx.z*blockDim.z + threadIdx.z;
	const int i = x + y*rb + z*rb*wb;
	const int ix_1 = (x - 1) + y *rb + z*rb*wb;
	const int iy_1 = x + (y - 1)*rb + z*rb*wb;
	const int iz_1 = x + y*rb + (z - 1)*rb*wb;
	const int iz0 = x + y*rb + 0 * rb*wb;


	if ((x < rb) && (y < wb) && (z == jb - 1))
	{
		er[i] = er[iz0];
		ew[i] = ew[iz0];
	}
	if (x == sr&&y == sw&&z == sj)
		er[i] += S;
	if ((x < rb - 1) && ((y == 0) || (y == wb - 1)) && (z == 0))
	{
		sumnum[x + (y / (wb - 1))*rb] = 0;
	}
}
__global__ void HKernel(const float *er, const float*ew, const float*ej, float*hr, float*hw, float*hj, const float da, const float db, const int rb, const int wb, const int jb, const float dr, const float dw, const float dj)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	const int z = blockIdx.z*blockDim.z + threadIdx.z;
	const int i = x + y*rb + z*rb*wb;
	const int i0 = x + y*rb + 0 * rb*wb;
	const int ix1 = (x + 1) + y *rb + z*rb*wb;
	const int iy1 = x + (y + 1)*rb + z*rb*wb;
	const int iz1 = x + y*rb + (z + 1)*rb*wb;

	if ((x > 0) && (x < rb - 1) && (y < wb - 1) && (z < jb - 1))
		hr[i] = da*hr[i] - db / ((x * dr + EARTH_RD)*sin((y + 0.5)*dw))*((sin((y + 1)*dw)*ej[iy1] - sin(y * dw)*ej[i]) / dw - (ew[iz1] - ew[i]) / dj);
	if ((x < rb - 1) && (y > 0) && (y < wb - 1) && (z < jb - 1))
		hw[i] = da*hw[i] - db / (((x + 0.5) * dr + EARTH_RD))*((er[iz1] - er[i]) / (sin(y * dw)*dj) - ((x + 1 + EARTH_RD / dr)*ej[ix1] - (x + EARTH_RD / dr)*ej[i]));
	if ((x < rb - 1) && (y < wb - 1) && (z < jb))
		hj[i] = da*hj[i] - db / (((x + 0.5)*dr + EARTH_RD))*(((x + 1 + EARTH_RD / dr)*ew[ix1] - (x + EARTH_RD / dr)*ew[i]) - (er[iy1] - er[i]) / dw);

}


int main()
{
	/////////////////////////////////////////////////////////////////////////////
	//��������
	const int re = RE, we = WE, je = JE;
	const int rb = re + 1, wb = we + 1, jb = je + 1;
	//����x��y��z��ĵ�Ԫ����
	const double dr = EARTH_HD / re, dw = M_PI / we, dj = 2 * M_PI / je;
	//Ԫ���ߴ磨m��
	const double dt = dr / (2 * LIGHT_SPEED);
	double fs = 1.0 / dt;
	const int sr = 0, sw = we / 2, sj = je / 2;
	const int on = 5;
	const int or [] = { 2 }, ow[] = { sw / 2 ,sw / 2,sw / 2,sw / 2,sw / 2 }, oj[] = { sj / 2 + 10,sj / 2 + 20,sj / 2 + 30,sj / 2 + 40,sj / 2 + 50 };
	//�۲��
	const double totalSimulatedTime = TOTALSIMULATERTIME;
	const int maximumIteration = (int)(totalSimulatedTime / dt) + 1;
	//const int samplingPoint = maximumIteration/(10*(FREQUENCY / FREQUENCYRESOLUTION));
	const int samplingPoint = LIGHT_SPEED / (5 * FREQUENCY *dr);
	int allocatedBytes = 0;
	//һ�����ٷ����ֽ����ļ�����
	int iteration = 0;
	//�������������Ѿ������˶��ٴ�ʱ��
	float stimulus = 0.0;
	//����ʱ�䲽�ļ���ֵ
	float currentSimulatedTime = 0.0;
	//����Ϊ��λ��ʱ�佫������ģ��
	time_t startTime, nowTime, lastTime = 0;
	const float tau = (float)(0.5 / FREQUENCY / dt);


	float *ca;// = (2 * EPSILON_0*EPSILON_R - SIGMA*dt) / (2 * EPSILON_0*EPSILON_R + SIGMA*dt);
	float *cb;//= (2 * dt) / (2 * EPSILON_0*EPSILON_R + SIGMA*dt);
	float da = (2 * MU_0*MU_R - SIGMA_M * dt) / (2 * MU_0*MU_R + SIGMA_M*dt);
	float db = 2 * dt / (2 * MU_0*MU_R + SIGMA_M*dt);
	//��ά����ָ��
	float *er, *ew, *ej;
	float *hr, *hw, *hj;
	float *sumnum;
	float sigma[100];
#ifdef KNEE
	string filesite = "./knee";
#elif AVG
	string filesite = "./avg";
	float sigma_avg[100] = { -13.82032, -13.66546, -13.40335, -13.17205, -12.99382, -12.84368, -12.70918, -12.58078, -12.4639, -12.34632,
		-12.23973, -12.13118, -12.03225, -11.93083, -11.83736, -11.74204, -11.65429, -11.5665,0 - 11.48464, -11.40081,
		-11.32333, -11.24440, -11.17181, -11.09722, -11.02856, -10.95733, -10.89176, -10.82312, -10.75983, -10.69290,
		-10.62880, -10.56880, -10.51152, -10.45234, -10.38773, -10.31917, -10.25097, -10.17755, -10.11117, -10.04006,
		-9.969597, -9.882136, -9.819363, -9.746188, -9.689247, -9.628824, -9.587276, -9.555107, -9.534937, -9.507883,
		-9.481722, -9.464875, -9.441254, -9.408667, -9.377145, -9.292050, -9.220743, -9.104745, -9.013201, -8.864476,
		-8.753869, -8.573321, -8.446165, -8.238668, -8.098838, -7.873779, -7.726281, -7.496615, -7.34718, -7.170000,
		-7.023970, -6.847879, -6.722940, -6.546849, -6.370757, -6.245819, -6.120880, -6.02397, -5.92706, -5.833638,
		-5.756798, -5.662242, -5.584637, -5.486151, -5.405922, -5.287574, -5.194666, -5.050842, -4.941185, -4.766291,
		-4.641053, -4.433463, -4.293188, -4.043739, -3.886141, -3.576347, -3.397373, -3.013649, -2.813157, -2.613600
	};
#else
	string filesite = "./nothing";
#endif
	string filename = "/matlab";
	string fileform = ".txt";
	ofstream FileStream[on];
	//ָ��txt�ļ���ָ��
	for (int rr = 0; rr < on; rr++)
	{
		FileStream[rr].open(filesite + filename + to_string(rr) + fileform);
	}

	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(rb, TX), divUp(wb, TY), divUp(jb, TZ));

	//udaSetDevice(0);//���ж��GPUʱĬ��ʹ�õ�0��
	hipMallocManaged(&er, rb*wb*jb * sizeof(float));
	hipMallocManaged(&ew, rb*wb*jb * sizeof(float));
	hipMallocManaged(&ej, rb*wb*jb * sizeof(float));

	hipMallocManaged(&hr, rb*wb*jb * sizeof(float));
	hipMallocManaged(&hw, rb*wb*jb * sizeof(float));
	hipMallocManaged(&hj, rb*wb*jb * sizeof(float));

	hipMallocManaged(&sumnum, rb * 2 * sizeof(float));
	hipMallocManaged(&ca, rb * 3 * sizeof(float));
	hipMallocManaged(&cb, rb * 3 * sizeof(float));

	for (int i = 0; i < rb*wb*jb; i++)
	{
		er[i] = 0;
		ew[i] = 0;
		ej[i] = 0;

		hr[i] = 0;
		hw[i] = 0;
		hj[i] = 0;

	}
	for (int i = 0; i < 2 * rb; i++)
	{
		sumnum[i] = 0;
	}
	//knee
#ifdef KNEE
	for (int h = 0; h < 100; h++)
	{
		if(h<=55)
			sigma[h] = 5.6e-10*exp((h - 55) / 8.3) ;
		else
			igma[h] = 5.6e-10*exp((h - 55) / 2.9);
	}

#elif AVG
	for (int i = 0; i < 100; i++)
	{
		sigma[i] = pow(10, sigma_avg[i]);
	}
#else
	for (int i = 0; i < 100; i++)
	{
		sigma[i] = 0;
	}
#endif
	for (int i = 0; i < re; i++)
	{
		//(2 * EPSILON_0*EPSILON_R - SIGMA*dt) / (2 * EPSILON_0*EPSILON_R + SIGMA*dt)
		ca[i] = (2 * EPSILON_0*EPSILON_R - sigma[2 + 4 * i] * dt) / (2 * EPSILON_0*EPSILON_R + sigma[2 + 4 * i] * dt);

		cb[i] = (2 * dt) / (2 * EPSILON_0*EPSILON_R + sigma[2 + 4 * i] * dt);
	}
	for (int i = 1; i < re; i++)
	{
		ca[i + rb] = (2 * EPSILON_0*EPSILON_R - sigma[4 * i] * dt) / (2 * EPSILON_0*EPSILON_R + sigma[4 * i] * dt);
		ca[i + 2 * rb] = (2 * EPSILON_0*EPSILON_R - sigma[4 * i] * dt) / (2 * EPSILON_0*EPSILON_R + sigma[4 * i] * dt);

		cb[i + rb] = (2 * dt) / (2 * EPSILON_0*EPSILON_R + sigma[4 * i] * dt);
		cb[i + 2 * rb] = (2 * dt) / (2 * EPSILON_0*EPSILON_R + sigma[4 * i] * dt);

	}

	time(&startTime);

	int usesec, allusesec, usedsec;
	for (iteration = 0; iteration < maximumIteration; iteration++)
	{



		EKernel1 << <gridSize, blockSize >> > (hj, rb, wb, jb, sumnum);
		//hipDeviceSynchronize();

		EKernel2 << <gridSize, blockSize >> > (er, ew, ej, hr, hw, hj, ca, cb, rb, wb, jb, dr, dw, dj, sumnum);

		stimulus = -(electric(iteration + 1, dt) - electric(iteration, dt))*dt / (EPSILON_0*EPSILON_R) / (sin(sw*dw)*dw*(EARTH_RD + sr*dr)*dj*(EARTH_RD + sr*dr)*dw*dr);
		//hipDeviceSynchronize();

		EKernel3 << <gridSize, blockSize >> > (er, ew, ej, sr, sw, sj, stimulus, sumnum, rb, wb, jb);

		//hipDeviceSynchronize();

		HKernel << <gridSize, blockSize >> > (er, ew, ej, hr, hw, hj, da, db, rb, wb, jb, dr, dw, dj);

		//hipDeviceSynchronize();
		//�˴����ӻ���840m����������⣬��1080ti��û��
		time(&nowTime);
		if (nowTime>lastTime)
		{
			currentSimulatedTime = dt*(double)iteration;
			//����ʱ��ģ���Ѿ����У�
			system("cls");
			//time(&nowTime);
			usedsec = nowTime - startTime;
			usesec = (int)(((double)(nowTime - startTime))*(maximumIteration - iteration) / (iteration + 1));
			allusesec = (int)(((double)(nowTime - startTime))*(maximumIteration) / (iteration + 1));
			//��ӡ����׼�����������͵�ǰģ��ʱ�䣺
			cout << iteration << " / " << maximumIteration << " " << currentSimulatedTime << "sec " << endl;
			cout << "use:" << usedsec / 3600 << "h " << (usedsec % 3600) / 60 << "m " << usedsec % 60 << "s   ";

			cout << "need:  " << usesec / 3600 << "h " << (usesec % 3600) / 60 << "m " << usesec % 60 << "s   ";
			cout << "all:  " << allusesec / 3600 << "h " << (allusesec % 3600) / 60 << "m " << allusesec % 60 << "s " << endl;


			cout << "speed:" << ((double)rb*wb*jb*iteration) / (nowTime - startTime) / 1.0e6 << " Mceil/s" << endl;
			lastTime = nowTime;
			/*	for (int k = 0; k < jb; k += 4)
			{
			for (int j = 0; j < wb; j += 4)
			{
			cout << er[or +j*rb + k*rb*wb] << " ";
			}
			cout << endl;
			}*/
		}


		if (!(iteration%samplingPoint))
		{
			hipDeviceSynchronize();
			for (int oo = 0; oo < on; oo++)
			{
				for (int r = 0; r < re; r++)
				{
					FileStream[oo] << er[r + ow[oo] * rb + oj[oo] * rb*wb] << " " <<
						ew[r + ow[oo] * rb + oj[oo] * rb*wb] << " " <<
						ej[r + ow[oo] * rb + oj[oo] * rb*wb] << " " <<
						hr[r + ow[oo] * rb + oj[oo] * rb*wb] << " " <<
						hw[r + ow[oo] * rb + oj[oo] * rb*wb] << " " <<
						hj[r + ow[oo] * rb + oj[oo] * rb*wb] << " ";
				}
				FileStream[oo] << endl;
			}

		}
	}
	for (int rr = 0; rr < on; rr++)
		FileStream[rr].close();
	cout << "all ok" << endl;
	hipFree(er);
	hipFree(ew);
	hipFree(ej);
	hipFree(hr);
	hipFree(hw);
	hipFree(hj);
	hipFree(sumnum);
	hipFree(ca);
	hipFree(cb);

	return 0;
}
